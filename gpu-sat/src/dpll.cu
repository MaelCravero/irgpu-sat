#include "hip/hip_runtime.h"
#include <optional>
#include <vector>
#include <iostream>

#include "cnf.cuh"
#include "dpll.cuh"
#include "utils.cuh"

namespace host
{
    namespace
    {
        void backjump(std::vector<term_val>& constants, size_t& pos)
        {
            while (pos >= 1 && constants[pos - 1] > 0)
            {
                constants[pos - 1] = 0;
                pos--;
            }

            if (pos)
                constants[pos - 1] *= -1;
        }

        solution calculate_solution(std::vector<term_val>& constants)
        {
            solution sol;

            int i = 1;
            for (auto sign : constants)
                sol.emplace_back(i * sign);

            return sol;
        }

    } // namespace

    std::optional<solution> dpll_solve(term_val* cnf_matrix, size_t nb_var,
                                       size_t nb_clause)
    {
        // FIXME error codes should be checked (but i'm lazy)

        std::vector<term_val> constants(nb_var);

        int nb_blocks = (nb_clause / 1024) + 1;

        size_t constant_pos = 0;
        for (;;) {

            std::cout << "\n" << "curent constants:\n";
            for (auto tv : constants)
                std::cout << (int)tv << " ";
            std::cout << "\n";

            term_val* local_cnf;
            hipMalloc(&local_cnf, nb_var * nb_clause * sizeof(term_val));
            hipMemcpy(local_cnf, cnf_matrix,
                       nb_var * nb_clause * sizeof(term_val),
                       hipMemcpyHostToDevice); // cnf seems to be on host

            size_t clause_size = nb_clause * sizeof(bool);
            bool* mask;
            hipMalloc(&mask, clause_size);

            auto cur_constant = 0;
            if (constant_pos)
            {
                cur_constant = constants[constant_pos - 1];
                constants[constant_pos - 1] = 0;
            }

            auto dev_constants = utils::init_from(constants);

            if (constant_pos)
            {
                constants[constant_pos - 1] = cur_constant;
            }

            device::simplify<<<nb_blocks, 1024>>>(local_cnf, nb_var, nb_clause,
                                                  dev_constants, mask);

            term_val* host_local_cnf = (term_val*)malloc(nb_clause * sizeof(term_val) * nb_var);
            hipMemcpy(host_local_cnf, local_cnf, nb_clause * sizeof(term_val) * nb_var, hipMemcpyDeviceToHost);

            bool* host_mask = (bool*)malloc(clause_size);
            hipMemcpy(host_mask, mask, clause_size, hipMemcpyDeviceToHost);

            std::cout << "cnf after simplify";
            for (int i = 0; i < nb_var * nb_clause; i++)
            {
                if (i % nb_var == 0)
                    std::cout << "\nMask = " << std::boolalpha << host_mask[i / nb_var] << "   ";
                std::cout << (int)host_local_cnf[i] << " ";
            }
            std::cout << "\n";

            free(host_local_cnf);
            free(host_mask);

            hipFree(dev_constants);
            bool conflict = false;

            if (constant_pos)
            {
                bool* results;
                hipMalloc(&results, nb_clause * sizeof(bool));

                device::check_conflict<<<nb_blocks, 1024>>>(
                    local_cnf, nb_var, nb_clause, constant_pos - 1,
                    constants[constant_pos - 1], results, mask);

                bool* host_res = (bool*)malloc(clause_size);
                hipMemcpy(host_res, results, clause_size, hipMemcpyDeviceToHost);

                for (auto i = 0; i < nb_clause; i++)
                {
                    std::cout << "conflict: " << std::boolalpha << host_res[i] << "\n";
                    if (host_res[i])
                        conflict = true;
                }

                free(host_res);
                hipFree(results);
            }

            hipFree(local_cnf);
            hipFree(mask);

            if (conflict)
            {
                backjump(constants, constant_pos);

                if (!constant_pos)
                    return {};
            }
            else if (constant_pos < nb_var)
            {
                constants[constant_pos++] = -1;
            }
            else
                break;
        }

        return {calculate_solution(constants)};
    }
} // namespace host

namespace device
{
    __global__ void check_conflict(term_val* cnf_matrix, size_t nb_var,
                                   size_t nb_clause, size_t constant_pos,
                                   term_val constant_sign, bool* results,
                                   bool* mask)
    {
        auto x = utils::x_idx();

        if (x >= nb_clause)
            return;

        results[x] = false;
        if (mask[x])
            return;

        bool conflict = cnf_matrix[x * nb_var + constant_pos] == -constant_sign;

        if (!conflict)
            return;

        int vars_in_clause = 0;
        for (auto i = x * nb_var; i < (x + 1) * nb_var; i++)
        {
            if (cnf_matrix[i])
            {
                if (!vars_in_clause)
                    vars_in_clause++;
                else
                {
                    vars_in_clause++;
                    break;
                }
            }
        }

        if (vars_in_clause == 1)
            results[x] = true;
    }

    __global__ void simplify(term_val* cnf_matrix, size_t nb_var,
                             size_t nb_clause, term_val* constants, bool* mask)
    {
        auto x = utils::x_idx();

        if (x >= nb_clause)
            return;

        mask[x] = false;

        for (auto i = 0; i < nb_var; i++)
        {
            auto pos = x * nb_var + i;

            if (cnf_matrix[pos] && cnf_matrix[pos] == -constants[i])
                cnf_matrix[pos] = 0; // The term can be removed

            else if (cnf_matrix[pos] && cnf_matrix[pos] == constants[i])
            {
                mask[x] = true; // The clause is true
                return;
            }
        }
    }

} // namespace device
